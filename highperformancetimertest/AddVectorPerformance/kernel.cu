#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "string"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "../highperformancetimertest/highperformancetimer.h"

using namespace std;

typedef int ArrayType_T;


int main(int argc, char * argv[])
{
	srand((unsigned)time(NULL));

	bool initialize(ArrayType_T ** a, ArrayType_T  ** b, ArrayType_T ** c, int size);
	void clearMem(ArrayType_T * a, ArrayType_T * b, ArrayType_T * c);
	void assign(ArrayType_T * a, ArrayType_T  * b, ArrayType_T * c, int size);
	void addVector(ArrayType_T *a, ArrayType_T *b, ArrayType_T *c, int size);
	void cudaMal(ArrayType_T *a, ArrayType_T *b, ArrayType_T *c, int size);


	int size = 1000;
	double accumulatedTime = 0.0;
	HighPrecisionTime htp;

	ArrayType_T * a = nullptr;
	ArrayType_T * b = nullptr;
	ArrayType_T * c = nullptr;


	if (argc > 1)
	{
		size = stoi(argv[1]);
	}

	cout << argv[0] << endl;
	cout << endl;
	cout << "Array size will be " << size << endl;


	try
	{
		if (!initialize(&a, &b, &c, size))
			throw("CPU memory allocation error ");
		cout << "CPU memory has been allocated" << endl;
		assign(a, b, c, size);

		cout << *a << endl;
		cout << *b << endl;
		cout << *c << endl;

		accumulatedTime = 0.0;

		for (int i = 0; i < 100; i++)
		{
			htp.TimeSinceLastCall();
			addVector(a, b, c, size);
			accumulatedTime += htp.TimeSinceLastCall();
		}
		cout << "Average time: to compute c = a+b: " << accumulatedTime / 100.0 << endl;

		
	
	}
	catch (char * errMessage)
	{
		cout << "An exception occured " << endl;
		cout << errMessage << endl;
	}
	clearMem(a, b, c);

	system("pause");


}


bool initialize(ArrayType_T ** a, ArrayType_T ** b, ArrayType_T ** c, int size)
{
	bool retVal = true;

	*a = (ArrayType_T*)malloc(size * sizeof(ArrayType_T));
	*b = (ArrayType_T*)malloc(size * sizeof(ArrayType_T));
	*c = (ArrayType_T*)malloc(size * sizeof(ArrayType_T));

	if (*a == nullptr || *b == nullptr || *c == nullptr)
	{
		retVal = false;
	}

	return retVal;
}

void assign(ArrayType_T * a, ArrayType_T  * b, ArrayType_T * c, int size)
{
	cout << "Starting loop" << endl;
	for (int i = 0; i < size; i++)
	{
		a[i] = rand() % size;
		b[i] = rand() % size;
		c[i] = 0;

	}

}

void clearMem(ArrayType_T * a, ArrayType_T * b, ArrayType_T * c)
{
	if (a != nullptr)
	{
		free(a);
	}
	if (b != nullptr)
	{
		free(b);
	}
	if (c != nullptr)
	{
		free(c);
	}

}

void addVector(ArrayType_T *a, ArrayType_T *b, ArrayType_T *c, int size)
{
	for (int i = 0; i < size; i++)
	{
		c[i] = a[i] + b[i];
	}
}
// Helper function for using CUDA to add vectors in parallel.
void cudaMal(ArrayType_T *a, ArrayType_T *b, ArrayType_T *c, int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);

	try
	{
		// Allocate GPU buffers for three vectors (two inputs A and B, one output C)    
		if (cudaStatus != hipSuccess) {
			throw("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		}


		cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int)); //mallocs in devA memory
		if (cudaStatus != hipSuccess) {
			throw("hipSetDevice a has failed");
		}

		cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int)); //mallocs in devB memory
		if (cudaStatus != hipSuccess) {
			throw("hipSetDevice b has failed");
		}


		cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int)); //mallocs in devC memory
		if (cudaStatus != hipSuccess) {
			throw("hipSetDevice c has failed");
		}
																	 // Copy input vectors from memory to GPU buffers. Not including C because
		cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			throw("hipMemcpy a has failed");
		}
		
		cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			throw("hipMemcpy b has failed");
		}

	}
	catch(char * errMessage)
	{
		cout << "An exception occured " << endl;
		cout << errMessage << endl;
	}

	hipFree(dev_c); //cleaning up after yourself
	hipFree(dev_a);
	hipFree(dev_b);
}
