#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "string"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "../highperformancetimertest/highperformancetimer.h"

using namespace std;

typedef int ArrayType_T;

int main(int argc, char * argv[])
{
	srand((unsigned)time(NULL));

	bool initialize(ArrayType_T ** a, ArrayType_T  ** b, ArrayType_T ** c, int size);
	void clearMem(ArrayType_T * a, ArrayType_T * b, ArrayType_T * c);
	void assign(ArrayType_T * a, ArrayType_T  * b, ArrayType_T * c, int size);
	void addVector(ArrayType_T *a, ArrayType_T *b, ArrayType_T *c, int size);


	int size = 1000;
	
	ArrayType_T * a = nullptr;
	ArrayType_T * b = nullptr;
	ArrayType_T * c = nullptr;

	double accumulatedTime = 0.0;

	//HighPrecisionTime htp;


	try
	{

		if (!initialize(&a, &b, &c, size))
			throw("CPU memory allocation error ");
		cout << "CPU memory has been allocated" << endl;

		//accumulatedTime = 0.0;

		//for (int i = 0; i < 100; i++)
		//{
			//htp.TimeSinceLastCall();
			//addVector(a, b, c, size);
			//accumulatedTime += htp.TimeSinceLastCall();
		//}

	}
	catch (char * errMessage)
	{
		cout << "An exception occured " << endl;
		cout << errMessage << endl;
	}
	cout << argc << endl;

	std::stoi(argv[1]);

	if (argc > 1)
	{
		size = stoi(argv[1]);
	}

	cout << argv[0] << endl;
	cout << endl;
	cout << "Array size will be " << size << endl;

	assign(a, b, c, size);

	cout << *a << endl;
	cout << *b << endl;
	cout << *c << endl;

	addVector(a, b, c, size);

	cout << "Adding the vectors by taking the random int of a and b and adding them and placing them into c" << endl;
	cout << *a << endl;
	cout << *b << endl;
	cout << *c << endl;

	clearMem(a, b, c);

	system("pause");


}


bool initialize(ArrayType_T ** a, ArrayType_T ** b, ArrayType_T ** c, int size)
{
	bool retVal = true;

	*a = (ArrayType_T*)malloc(size * sizeof(ArrayType_T));
	*b = (ArrayType_T*)malloc(size * sizeof(ArrayType_T));
	*c = (ArrayType_T*)malloc(size * sizeof(ArrayType_T));

	if (*a == nullptr || *b == nullptr || *c == nullptr)
	{
		retVal = false;
	}

	return retVal;
}

void assign(ArrayType_T * a, ArrayType_T  * b, ArrayType_T * c, int size)
{
	cout << "Starting loop" << endl;
	for (int i = 0; i < size; i++)
	{
		a[i] = rand() % size;
		b[i] = rand() % size;
		c[i] = 0;

	}

}

void clearMem(ArrayType_T * a, ArrayType_T * b, ArrayType_T * c)
{
	if (a != nullptr)
	{
		free(a);
	}
	if (b != nullptr)
	{
		free(b);
	}
	if (c != nullptr)
	{
		free(c);
	}

}

void addVector(ArrayType_T *a, ArrayType_T *b, ArrayType_T *c, int size)
{
	for (int i = 0; i < size; i++)
	{
		c[i] = a[i] + b[i];
	}
}